#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#define K 500
#define N 40000000
#define k 10

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }
__device__ int table[N][k];


__global__ void buildTable(const char *text, int *pos, int text_size, int depth)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int level = 1 << depth;
    if(idx >= text_size) {
        return;
    }
	if(depth == 0) {
		if( text[idx]  == '\n') {
			table[idx][0] = 0;
		} else {
			table[idx][0] = 1;
		}
	} else {
		if( idx < text_size/level ) {
			if( table[idx*2][depth-1]==1 && table[idx*2+1][depth-1]==1 ) {
				table[idx][depth] = 1;
			}
		}
	}
}


__global__ void countPos(const char *text, int *pos, int text_size)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= text_size) {
        return ;
    }
	
	if(table[idx][0] == 0) {
		// if the it represents newline
		// return to 
		pos[idx] = 0;
	} else {
		int length = 0;
		int index = idx;
		int add_length = 1;
		int depth = 0;
		while(1) {
			if(index <= 0) {
				break;
			}
			if (index%2 == 0) {
				// Tf the child is the left child, will have to travel to the parents' left node.
				// That is, the length will have to be added by add_length;
				index = (index-1);
				length = length + add_length;
                //printf("%2: len=%d, index=%d, add=%d, depth=%d\n", length, index, add_length, depth);
			}
			if(table[(index-1)/2][depth+1] == 1) {
				add_length = add_length*2;
				depth = depth+1;
				index = (index-1)/2;
                //printf("checktable: len=%d, index=%d, add=%d, depth=%d\n", length, index, add_length, depth);
			} else {
                //printf("else1: len=%d, index=%d, add=%d, depth=%d\n", length, index, add_length, depth);
				break;
			}
		}
		while(depth >= 0 && index >= 0 && add_length > 0) {
			if(table[index][depth] == 1) {
				// the parent is 0 and have to do to left child's left node
				index = index*2 - 1;
				length = length + add_length;
				depth = depth - 1;
				add_length = add_length/2;
                //printf("if1: len=%d, index=%d, add=%d, depth=%d\n", length, index, add_length, depth);
			} else {
				// table[idx][depth] == 0 and have to go to the right child
				index = index*2 + 1;
				depth = depth - 1;
				add_length = add_length/2;
                //printf("if2: len=%d, index=%d, add=%d, depth=%d\n", length, index, add_length, depth);
			}
		}
		pos[idx] = length;
	}
	
}

__global__ void test(const char *text, int *pos, int text_size)
{
	int count=0;
	for(int i=0; i<text_size; i++) {
		if(text[i] == '\n')
			count = 0;
		else
			count = count+1;
		pos[i] = count;
	}
}

__global__ void printTable()
{
	printf("print table\n");
	for(int i=0; i<6; i++) {
		for(int j=0; j<100; j++)
			printf("%d", table[j][i]);
		printf("\n");
	}
}
void CountPosition(const char *text, int *pos, int text_size)
{
	int count = text_size/2;
	printf("text size: %d\n", text_size);
    for(int depth=0; depth<k; depth++) {
	    buildTable<<<40000, 1024>>>(text, pos, text_size, depth);
        hipDeviceSynchronize();
    }
	countPos<<<40000, 1024>>>(text, pos, text_size);
	//test<<<1,1>>>(text, pos, text_size);
}

struct is_one {
    __device__ __host__ bool operator()(const int &check) {
        return check == 1;
    }
};

int ExtractHead(const int *pos, int *head, int text_size)
{
	printf("check\n");
    int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer+text_size);
    
    printf("Extract Head\n");
	// TODO
    nhead = thrust::count(thrust::device, pos_d, pos_d+text_size, 1);
    thrust::device_vector<int> seq(text_size);
    for(int i=0; i<text_size; i++) {
        seq[i] = i;
        printf("In seq:%d\n", i);
    }
    thrust::copy_if(seq.begin(), seq.end(), pos_d, head_d, is_one());
    printf("Extract Head\n");


	hipFree(buffer);
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
}
